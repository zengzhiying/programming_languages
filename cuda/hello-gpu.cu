
#include <hip/hip_runtime.h>
#include <stdio.h>

/**
 * 第一个gpu程序: Hello GPU.
 * cuda运行方式: nvcc -arch=sm_70 -o hello-gpu hello-gpu.cu -run
 * 然后编译会生成: hello-gpu可执行文件, 之后可直接执行
 */

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * Refactor the `helloGPU` definition to be a kernel
 * that can be launched on the GPU. Update its message
 * to read "Hello from the GPU!"
 */
__global__ void helloGPU()
{
  printf("Hello also from the GPU.\n");
}

int main()
{

  helloCPU();

  /*
   * Refactor this call to `helloGPU` so that it launches
   * as a kernel on the GPU.
   */
  helloGPU<<<1, 1>>>();

  /*
   * Add code below to synchronize on the completion of the
   * `helloGPU` kernel completion before continuing the CPU
   * thread.
   */
  hipDeviceSynchronize();
}
