
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

/**
 * cpu实现
 */
void loop(int N)
{
  for (int i = 0; i < N; ++i)
  {
    printf("CPU This is iteration number %d\n", i);
  }
}
/**
 * gpu实现
 */
__global__ void loop()
{
    // 这里循环均适用block和线程替代来实现并行计算
    // blockDim表示一个块中的线程数量, 通过blockIdx.x * blockDim * x + threadIdx.x来实现循环区间索引.
    printf("GPU This is iteration number %d\n", blockIdx.x * blockDim.x + threadIdx.x);
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, only use 1 block of threads.
   */
  int N = 10;
  loop(N);
  loop<<<2, 10>>>();
  hipDeviceSynchronize();
}
