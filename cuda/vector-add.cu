
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

/**
 * cuda实现向量相加计算
 */

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

/**
 * cpu实现向量加法
 */
void addVectorsInto(float *result, float *a, float *b, int N)
{
  for(int i = 0; i < N; ++i)
  {
    result[i] = a[i] + b[i];
  }
}

/**
 * gpu实现向量加法
 */
__global__ void addVectorsIntoWithCuda(float *result, float *a, float *b, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  
  for(int i = idx; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  //a = (float *)malloc(size);
  //b = (float *)malloc(size);
  //c = (float *)malloc(size);
  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  // addVectorsInto(c, a, b, N);
  addVectorsIntoWithCuda<<<128, 1024>>>(c, a, b, N);
  checkCuda(hipDeviceSynchronize());

  checkElementsAre(7, c, N);

  //free(a);
  //free(b);
  //free(c);
  hipFree(a);
  hipFree(b);
  hipFree(c);
}
