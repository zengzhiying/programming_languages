
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__ void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  size_t threads_per_block = 2048;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();
  
  // 错误处理部分代码, 可以确定在gpu中运行的代码是否发生错误.
  hipError_t err;
  err = hipGetLastError();
  if(err != hipSuccess) {
      printf("Error: %s\n", hipGetErrorString(err));
  }
  

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);

  return 0;
}
