
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * 最简单的基于cuda在gpu并行运行的函数
 */
__global__ void firstParallel()
{
  printf("This should be running in parallel.\n");
}

int main()
{
  firstParallel<<<1, 10>>>();
  hipDeviceSynchronize();

  return 0;
}
